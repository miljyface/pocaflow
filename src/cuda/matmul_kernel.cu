#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define TILE_SIZE 32
#define BLOCK_SIZE 256

// Optimized SGEMM kernel with shared memory tiling and vectorized loads
__global__ void matmul_kernel_optimized(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    int M, int N, int K)
{
    __shared__ float As[TILE_SIZE][TILE_SIZE + 1]; // +1 to avoid bank conflicts
    __shared__ float Bs[TILE_SIZE][TILE_SIZE + 1];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float sum = 0.0f;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load tiles into shared memory with bounds checking
        if (row < M && t * TILE_SIZE + tx < K)
            As[ty][tx] = A[row * K + t * TILE_SIZE + tx];
        else
            As[ty][tx] = 0.0f;

        if (col < N && t * TILE_SIZE + ty < K)
            Bs[ty][tx] = B[(t * TILE_SIZE + ty) * N + col];
        else
            Bs[ty][tx] = 0.0f;

        __syncthreads();

        // Compute partial dot product
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    // Write result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

// C wrapper for Rust FFI
extern "C" {
    void launch_matmul_kernel(
        const float* d_A,
        const float* d_B,
        float* d_C,
        int M, int N, int K,
        hipStream_t stream)
    {
        dim3 blockDim(TILE_SIZE, TILE_SIZE);
        dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
        
        matmul_kernel_optimized<<<gridDim, blockDim, 0, stream>>>(d_A, d_B, d_C, M, N, K);
    }
}
